#include "hip/hip_runtime.h"
/**
 * calculate pi
 */

#include <stdio.h>
#include <math.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <vector>

#include "types.h"
#include "CNN/layer_t.h"

/**
 * CUDA Kernel Device code
 *
 */
/*****************************************************************************/

__global__ void training2(case_t *d_cases, long int batchSize) {

	int index = (blockDim.x * blockIdx.x) + threadIdx.x;

	printf("Index: %i, Cases: %f", index, sizeof(d_cases));


	__syncthreads();

}

/******************************************************************************
 * Host main routine
 */
std::vector<std::vector<layer_t*>> cuda_training(std::vector<case_t> cases, int batchSize,
		std::vector<std::vector<layer_t*>> slaves){

	int blocksPerGrid, threadsPerBlock, i, size;
	int totalThreads;
	case_t *h_cases, *d_cases;
	int *h_batchSize, *d_batchSize;
	std::vector<std::vector<layer_t*>> *h_slaves, *d_slaves;

	// Get device info
	int dev = 0;
	hipSetDevice(dev);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);

	blocksPerGrid = deviceProp.multiProcessorCount;
	int cudaCores = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
	threadsPerBlock = deviceProp.maxThreadsPerBlock;
	totalThreads = blocksPerGrid * threadsPerBlock;

	hipError_t err = hipSuccess;

	h_cases = &cases[0];
	size = sizeof(case_t)*cases.size();

	if (h_cases == NULL) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	// copy vector to array
	copy(cases.begin(), cases.end(), h_cases);


	err = hipMalloc((void **) &d_cases, size);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_cases, h_cases, size,
			hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to copy vector C from device to host (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Lanzar KERNEL

	printf("CUDA kernel launch with %d blocks of %d threads. Total: %i\n",
			blocksPerGrid, threadsPerBlock, totalThreads);
	training2<<<blocksPerGrid, threadsPerBlock>>>(d_cases, batchSize);
	err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(h_cases, d_cases, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to copy vector C from device to host (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_cases);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to free device vector C (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Free host memory

	free(h_cases);
	err = hipDeviceReset();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to deinitialize the device! error=%s\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	return slaves;
}

