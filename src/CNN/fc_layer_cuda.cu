#include "hip/hip_runtime.h"
#pragma once
#include <stdio.h>
#include <math.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <vector>
#include <algorithm>   // std::min

#include <math.h>
#include <float.h>
#include <string.h>

#include "gradient_t.h"
#include "layer_t.h"
#include "optimization_method.h"

#include "fc_layer_cuda.cuh"

void activate2cuda(tensor_t<float> in, tensor_t<float> out);


fc_layer_cuda_t::fc_layer_cuda_t(tdsize in_size, int out_size) :
		in(in_size.x, in_size.y, in_size.z), out(out_size, 1, 1), grads_in(
				in_size.x, in_size.y, in_size.z), weights(
				in_size.x * in_size.y * in_size.z, out_size, 1) {
	input = std::vector<float>(out_size);
	gradients = std::vector<gradient_t>(out_size);

	int maxval = in_size.x * in_size.y * in_size.z;

	for (int i = 0; i < out_size; i++)
		for (int h = 0; h < in_size.x * in_size.y * in_size.z; h++)
			weights(h, i, 0) = 2.19722f / maxval * rand() / float(RAND_MAX);
	// 2.19722f = f^-1(0.9) => x where [1 / (1 + exp(-x) ) = 0.9]
}

float activator_function(float x) {
	//return tanhf( x );
	float sig = 1.0f / (1.0f + exp(-x));
	return sig;
}

float activator_derivative(float x) {
	//float t = tanhf( x );
	//return 1 - t * t;
	float sig = 1.0f / (1.0f + exp(-x));
	return sig * (1 - sig);
}

void fc_layer_cuda_t::activate(tensor_t<float>& in) {
	this->in = in;
	//activate();
	activate2cuda(in, out);
}

int fc_layer_cuda_t::map(point_t d) {
	return d.z * (in.size.x * in.size.y) + d.y * (in.size.x) + d.x;
}

void fc_layer_cuda_t::activate() {
	for (int n = 0; n < out.size.x; n++) {
		float inputv = 0;

		for (int i = 0; i < in.size.x; i++)
			for (int j = 0; j < in.size.y; j++)
				for (int z = 0; z < in.size.z; z++) {
					int m = map( { i, j, z });
					inputv += in(i, j, z) * weights(m, n, 0);
				}

		input[n] = inputv;

		out(n, 0, 0) = activator_function(inputv);
	}
}

void fc_layer_cuda_t::fix_weights() {
	for (int n = 0; n < out.size.x; n++) {
		gradient_t& grad = gradients[n];
		for (int i = 0; i < in.size.x; i++)
			for (int j = 0; j < in.size.y; j++)
				for (int z = 0; z < in.size.z; z++) {
					int m = map( { i, j, z });
					float& w = weights(m, n, 0);
					w = update_weight(w, grad, in(i, j, z));
				}

		update_gradient(grad);
	}
}

void fc_layer_cuda_t::calc_grads(tensor_t<float>& grad_next_layer) {
	memset(grads_in.data, 0,
			grads_in.size.x * grads_in.size.y * grads_in.size.z
					* sizeof(float));
	for (int n = 0; n < out.size.x; n++) {
		gradient_t& grad = gradients[n];
		grad.grad = grad_next_layer(n, 0, 0) * activator_derivative(input[n]);

		for (int i = 0; i < in.size.x; i++)
			for (int j = 0; j < in.size.y; j++)
				for (int z = 0; z < in.size.z; z++) {
					int m = map( { i, j, z });
					grads_in(i, j, z) += grad.grad * weights(m, n, 0);
				}
	}
}

void fc_layer_cuda_t::setWeights(tensor_t<float> newWeights) {
	weights = newWeights;
}
void fc_layer_cuda_t::updateWeights(tensor_t<float> newWeights) {
	weights = (weights + newWeights) / 2;
}

/**
 * CUDA Kernel Device code
 *
 */
/*****************************************************************************/

__global__ void activate_cuda(tensor_t<float> *d_in, tensor_t<float> *d_out) {

	int index = (blockDim.x * blockIdx.x) + threadIdx.x;

	printf("Index: %i, in size: (%i, %i, %i) \t", index, d_in->size.x,
			d_in->size.y, d_in->size.z);

}

/******************************************************************************
 * Host main routine
 */
void activate2cuda(tensor_t<float> in, tensor_t<float> out) {

	int blocksPerGrid, threadsPerBlock;
	int totalThreads;
	tensor_t<float> *h_in, *d_in;
	tensor_t<float> *h_out, *d_out;

	// Get device info
	int dev = 0;
	hipSetDevice(dev);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);

	blocksPerGrid = std::min(deviceProp.multiProcessorCount, 2);
	int cudaCores = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
	threadsPerBlock = deviceProp.maxThreadsPerBlock;
	totalThreads = blocksPerGrid * threadsPerBlock;

	hipError_t err = hipSuccess;

	h_in = &in;
	int in_mem_size = sizeof(in);

	// TODO remove
	printf("sizeof(in) , sizeof(out) = %lu * %lu \n", sizeof(in), sizeof(in));

	if (h_in == NULL) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **) &d_in, in_mem_size);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_in, h_in, in_mem_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to copy vector C from device to host (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy slaves to device

	h_out = &out;
	int out_mem_size = sizeof(out);

	if (h_in == NULL) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **) &d_out, out_mem_size);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_out, h_out, out_mem_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to copy vector C from device to host (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Lanzar KERNEL

	printf("CUDA kernel launch with %d blocks of %d threads. Total: %i\n",
			blocksPerGrid, threadsPerBlock, totalThreads);

	activate_cuda<<<blocksPerGrid, threadsPerBlock>>>(d_in, d_out);

	err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(h_out, d_out, out_mem_size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to copy vector C from device to host (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_out);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to free device vector C (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Free host memory

	//free(h_cases);
	err = hipDeviceReset();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to deinitialize the device! error=%s\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	// TODO remove
	printf("cuda out: ");
}

