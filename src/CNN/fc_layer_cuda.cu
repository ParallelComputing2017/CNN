#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <vector>
#include <algorithm>   // std::min

#include <math.h>
#include <float.h>
#include <string.h>

#include "gradient_t.h"
#include "layer_t.h"
#include "optimization_method.h"

#include "fc_layer_cuda.cuh"

#include "CUDA/utils.cuh"

void activate2cuda(tensor_t<float> in, tensor_t<float> weights,
		std::vector<float> &input, tensor_t<float> &out);

fc_layer_cuda_t::fc_layer_cuda_t(tdsize in_size, int out_size) :
		in(in_size.x, in_size.y, in_size.z), out(out_size, 1, 1), grads_in(
				in_size.x, in_size.y, in_size.z), weights(
				in_size.x * in_size.y * in_size.z, out_size, 1) {
	input = std::vector<float>(out_size);
	gradients = std::vector<gradient_t>(out_size);

	int maxval = in_size.x * in_size.y * in_size.z;

	for (int i = 0; i < out_size; i++)
		for (int h = 0; h < in_size.x * in_size.y * in_size.z; h++)
			weights(h, i, 0) = 2.19722f / maxval * rand() / float(RAND_MAX);
	// 2.19722f = f^-1(0.9) => x where [1 / (1 + exp(-x) ) = 0.9]
}

__host__ __device__ float activator_function(float x) {
	//return tanhf( x );
	float sig = 1.0f / (1.0f + exp(-x));
	return sig;
}

float activator_derivative(float x) {
	//float t = tanhf( x );
	//return 1 - t * t;
	float sig = 1.0f / (1.0f + exp(-x));
	return sig * (1 - sig);
}

__host__ void fc_layer_cuda_t::activate(tensor_t<float>& in) {
	this->in = in;
	//activate();

	// TODO
	printf("before activate");
	print_tensor(out);

	activate2cuda(in, weights, input, out);

	// TODO
	printf("\n after activate: ");
	print_tensor(out);
}

int fc_layer_cuda_t::map(point_t d) {
	return d.z * (in.getSize().x * in.getSize().y) + d.y * (in.getSize().x) + d.x;
}

void fc_layer_cuda_t::activate() {
	for (int n = 0; n < out.getSize().x; n++) {
		float inputv = 0;

		for (int i = 0; i < in.getSize().x; i++)
			for (int j = 0; j < in.getSize().y; j++)
				for (int z = 0; z < in.getSize().z; z++) {
					int m = map( { i, j, z });
					inputv += in(i, j, z) * weights(m, n, 0);
				}

		input[n] = inputv;

		out(n, 0, 0) = activator_function(inputv);
	}
}

void fc_layer_cuda_t::fix_weights() {
	for (int n = 0; n < out.getSize().x; n++) {
		gradient_t& grad = gradients[n];
		for (int i = 0; i < in.getSize().x; i++)
			for (int j = 0; j < in.getSize().y; j++)
				for (int z = 0; z < in.getSize().z; z++) {
					int m = map( { i, j, z });
					float& w = weights(m, n, 0);
					w = update_weight(w, grad, in(i, j, z));
				}

		update_gradient(grad);
	}
}

void fc_layer_cuda_t::calc_grads(tensor_t<float>& grad_next_layer) {
	memset(grads_in.data, 0,
			grads_in.getSize().x * grads_in.getSize().y * grads_in.getSize().z
					* sizeof(float));
	for (int n = 0; n < out.getSize().x; n++) {
		gradient_t& grad = gradients[n];
		grad.grad = grad_next_layer(n, 0, 0) * activator_derivative(input[n]);

		for (int i = 0; i < in.getSize().x; i++)
			for (int j = 0; j < in.getSize().y; j++)
				for (int z = 0; z < in.getSize().z; z++) {
					int m = map( { i, j, z });
					grads_in(i, j, z) += grad.grad * weights(m, n, 0);
				}
	}
}

void fc_layer_cuda_t::setWeights(tensor_t<float> newWeights) {
	weights = newWeights;
}
void fc_layer_cuda_t::updateWeights(tensor_t<float> newWeights) {
	weights = (weights + newWeights) / 2;
}

/**
 * CUDA Kernel Device code
 *
 */
/*****************************************************************************/

__device__ float& get(tensor_t<float> *t, int _x, int _y, int _z) {
	assert(_x >= 0 && _y >= 0 && _z >= 0);
	assert(_x < t->size.x && _y < t->size.y && _z < t->size.z);

	return t->data[_z * (t->size.x * t->size.y) + _y * (t->size.x) + _x];
}

__device__ void set(tensor_t<float> *t, int _x, int _y, int _z, float value) {
	assert(_x >= 0 && _y >= 0 && _z >= 0);
	assert(_x < t->size.x && _y < t->size.y && _z < t->size.z);

	t->data[_z * (t->size.x * t->size.y) + _y * (t->size.x) + _x] = value;
}

__global__ void activate_cuda(tensor_t<float> *d_in, tensor_t<float> *d_weights,
		float *d_input, tensor_t<float> *d_out) {

	int index = (blockDim.x * blockIdx.x) + threadIdx.x;

	printf("d_input 0: %f \n", *d_input);

	printf("d_out: %i, %i, %i \n", d_out->size.x, d_out->size.y, d_out->size.z);

	printf("d_weights: %i, %i, %i \n", d_weights->size.x, d_weights->size.y,
			d_weights->size.z);

	for (int n = 0; n < d_out->size.x; n++) {
		float inputv = 0;

		for (int i = 0; i < d_in->size.x; i++)
			for (int j = 0; j < d_in->size.y; j++)
				for (int z = 0; z < d_in->size.z; z++) {
					// map
					int m = z * (d_in->size.x * d_in->size.y)
							+ j * (d_in->size.x) + i;

					inputv += get(d_in, i, j, z) * get(d_weights, m, n, 0);

				}

		printf("inputv: %f \n", inputv);

		*(d_input + n) = inputv;

		set(d_out, n, 0, 0, activator_function(inputv));
	}

	printf("d_input 0: %f \n", *d_input);

}

/******************************************************************************
 * Host main routine
 */
void activate2cuda(tensor_t<float> in, tensor_t<float> weights,
		std::vector<float> &input, tensor_t<float> &out) {

	int blocksPerGrid, threadsPerBlock;
	int totalThreads;
	tensor_t<float> *h_in, *d_in;
	tensor_t<float> *h_weights, *d_weights;
	float *h_input, *d_input;
	tensor_t<float> *h_out, *d_out;

	// Get device info
	int dev = 0;
	hipSetDevice(dev);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);

	blocksPerGrid = std::min(deviceProp.multiProcessorCount, 1);
	int cudaCores = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
	threadsPerBlock = std::min(deviceProp.maxThreadsPerBlock, 1);
	totalThreads = blocksPerGrid * threadsPerBlock;

	h_in = &in;
	int in_mem_size = sizeof(in);

	// TODO remove
	printf("out[0,0,0]: %f, \n", out.get(0, 0, 0));

	printf("in: %i, %i, %i \n", in.getSize().x, in.getSize().y, in.getSize().z);

	if (h_in == NULL) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	hipMalloc((void **) &d_in, in_mem_size);
	cudaCheckError("hipMalloc IN tensor");

	hipMemcpy(d_in, h_in, in_mem_size, hipMemcpyHostToDevice);
	cudaCheckError("cudaMemcopy to device IN tensor");

	// IN DATA

	float *d_in_data;
	long in_data_size = sizeof(*d_in_data) * in.getSize().x * in.getSize().y * in.getSize().z;

	printf("sizeof(in)= %lu , in_data_size = %lu \n", sizeof(in), in_data_size);

	hipMalloc((void **) &d_in_data, in_data_size);
	cudaCheckError("hipMalloc IN tensor data");

	hipMemcpy(d_in_data, in.data, in_data_size, hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy to device IN tensor data");

	hipMemcpy(&(d_in->data), &d_in_data, sizeof(d_in->data),
			hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy Binding pointers of IN tensor data");

	// Copy weights

	h_weights = &weights;
	long weights_mem_size = sizeof(weights);

	printf("sizeof(weights) == weights_mem_size => %lu == %lu \n",
			sizeof(weights), weights_mem_size);

	if (h_weights == NULL) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	hipMalloc((void **) &d_weights, weights_mem_size);
	cudaCheckError("hipMalloc Weights tensor");

	hipMemcpy(d_weights, h_weights, weights_mem_size, hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy to device Weights tensor");

	// Weights DATA

	float *d_weights_data;
	long weights_data_size = sizeof(*d_weights_data) * weights.getSize().x
			* weights.getSize().y * weights.getSize().z;

	hipMalloc((void **) &d_weights_data, weights_data_size);
	cudaCheckError("hipMalloc Weights tensor data");

	hipMemcpy(d_weights_data, weights.data, weights_data_size,
			hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy to device Weights tensor data");

	hipMemcpy(&(d_weights->data), &d_weights_data, sizeof(d_weights->data),
			hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy Binding pointers of Weights tensor data");

	// Reserve input memory space

	h_input = &input[0];
	long input_mem_size = sizeof(input[0]) * input.size();

	printf("input_mem_size : %lu \n", input_mem_size);

	hipMalloc((void **) &d_input, input_mem_size);
	cudaCheckError("hipMalloc Input array");

	hipMemcpy(d_input, h_input, input_mem_size, hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy to device Input array");

	// Reserve memory space for OUT

	h_out = &out;
	int out_mem_size = sizeof(out);

	hipMalloc((void **) &d_out, out_mem_size);
	cudaCheckError("hipMalloc Out tensor");

	hipMemcpy(d_out, h_out, out_mem_size, hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy to device Out tensor");

	// Out DATA

	float *d_out_data;
	long out_data_size = sizeof(*d_out_data) * h_out->getSize().x * h_out->getSize().y
			* h_out->getSize().z;

	printf("out_data_size : %lu \n", out_data_size);

	hipMalloc((void **) &d_out_data, out_data_size);
	cudaCheckError("hipMalloc Out tensor data");

	hipMemcpy(d_out_data, h_out->data, out_data_size, hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy to device Out tensor data");

	hipMemcpy(&(d_out->data), &d_out_data, sizeof(d_out->data),
			hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy Binding pointers of Out tensor data");

	printf("h_out[0,0,0]: %f, \n", h_out->get(0, 0, 0));
	// TODO
	//printf("Tensor in: \n");
	//print_tensor(*h_in);

	// TODO remove
	printf("input size: %lu input 0: %f \n", input.size(), input[0]);

	printf("h_input 0: %f \n", *h_input);

	printf("out: %i, %i, %i \n", out.getSize().x, out.getSize().y, out.getSize().z);

	printf("h_out: %i, %i, %i \n", h_out->getSize().x, h_out->getSize().y, h_out->getSize().z);

	// Lanzar KERNEL

	printf("CUDA kernel launch with %d blocks of %d threads. Total: %i\n",
			blocksPerGrid, threadsPerBlock, totalThreads);

	activate_cuda<<<blocksPerGrid, threadsPerBlock>>>(d_in, d_weights, d_input,
			d_out);

	hipDeviceSynchronize();

	cudaCheckError("Launch kernel");

	// get input array

	hipMemcpy(h_input, d_input, input_mem_size, hipMemcpyDeviceToHost);
	cudaCheckError("hipMemcpy to host Input array");

	hipFree(d_input);
	cudaCheckError("hipFree Input array");

	// Get Out DATA

	hipMemcpy(h_out->data, d_out_data, out_data_size, hipMemcpyDeviceToHost);
	cudaCheckError("hipMemcpy to host Out tensor data");

	hipFree(d_out);
	cudaCheckError("hipFree Out tensor");

	// Free host memory

	hipDeviceReset();

	cudaCheckError("hipDeviceReset");
	// TODO remove
	printf("cuda out: %i, %i, %i \n", h_out->getSize().x, h_out->getSize().y,
			h_out->getSize().z);
	printf("cuda out[0,0,0]: %f, \n", h_out->get(0, 0, 0));
	print_tensor(*h_out);
}

