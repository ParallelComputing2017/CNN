#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <vector>
#include <algorithm>   // std::min

#include <math.h>
#include <float.h>
#include <string.h>

#include "gradient_t.h"
#include "layer_t.h"
#include "optimization_method.h"

#include "fc_layer_cuda.h"

#include "CUDA/utils.cuh"

void activate2cuda(tensor_t<float> in, tensor_t<float> weights,
		std::vector<float> &input, tensor_t<float> &out);

__host__ __device__ float activator_function(float x) {
	//return tanhf( x );
	float sig = 1.0f / (1.0f + exp(-x));
	return sig;
}

__host__ void fc_layer_cuda_t::activate(tensor_t<float>& in) {
	this->in = in;
	//activate();

	// TODO
	//printf("before activate");
	//print_tensor(this->out);

	activate2cuda(this->in, this->weights, this->input, this->out);

	// TODO
	//printf("\n after activate: ");
	//print_tensor(this->out);
}

/**
 * CUDA Kernel Device code
 *
 */
/*****************************************************************************/

__device__ float& get(tensor_t<float> *t, int _x, int _y, int _z) {
	assert(_x >= 0 && _y >= 0 && _z >= 0);
	assert(_x < t->size.x && _y < t->size.y && _z < t->size.z);

	return t->data[_z * (t->size.x * t->size.y) + _y * (t->size.x) + _x];
}

__device__ void set(tensor_t<float> *t, int _x, int _y, int _z, float value) {
	assert(_x >= 0 && _y >= 0 && _z >= 0);
	assert(_x < t->size.x && _y < t->size.y && _z < t->size.z);

	t->data[_z * (t->size.x * t->size.y) + _y * (t->size.x) + _x] = value;
}

__global__ void activate_cuda(tensor_t<float> *d_in, tensor_t<float> *d_weights,
		float *d_input, tensor_t<float> *d_out) {

	int index = (blockDim.x * blockIdx.x) + threadIdx.x;

	//printf("d_input 0: %f \n", *d_input);

	//printf("d_out: %i, %i, %i \n", d_out->size.x, d_out->size.y, d_out->size.z);

	//printf("d_weights: %i, %i, %i \n", d_weights->size.x, d_weights->size.y, d_weights->size.z);

	for (int n = 0; n < d_out->size.x; n++) {
		float inputv = 0;

		for (int i = 0; i < d_in->size.x; i++)
			for (int j = 0; j < d_in->size.y; j++)
				for (int z = 0; z < d_in->size.z; z++) {
					// map
					int m = z * (d_in->size.x * d_in->size.y)
							+ j * (d_in->size.x) + i;

					inputv += get(d_in, i, j, z) * get(d_weights, m, n, 0);

				}

		//printf("inputv: %f \n", inputv);

		*(d_input + n) = inputv;

		set(d_out, n, 0, 0, activator_function(inputv));
	}

	//printf("d_input 0: %f \n", *d_input);

}

/******************************************************************************
 * Host main routine
 */
void activate2cuda(tensor_t<float> in, tensor_t<float> weights,
		std::vector<float> &input, tensor_t<float> &out) {

	int blocksPerGrid, threadsPerBlock;
	int totalThreads;
	tensor_t<float> *h_in, *d_in;
	tensor_t<float> *h_weights, *d_weights;
	float *h_input, *d_input;
	tensor_t<float> *h_out, *d_out;

	// Get device info
	int dev = 0;
	hipSetDevice(dev);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);

	blocksPerGrid = std::min(deviceProp.multiProcessorCount, 1);
	int cudaCores = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
	threadsPerBlock = std::min(deviceProp.maxThreadsPerBlock, 1);
	totalThreads = blocksPerGrid * threadsPerBlock;

	h_in = &in;
	int in_mem_size = sizeof(in);

	// TODO remove
	//printf("out[0,0,0]: %f, \n", out.get(0, 0, 0));

	//printf("in: %i, %i, %i \n", in.getSize().x, in.getSize().y, in.getSize().z);

	if (h_in == NULL) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	hipMalloc((void **) &d_in, in_mem_size);
	cudaCheckError("hipMalloc IN tensor");

	hipMemcpy(d_in, h_in, in_mem_size, hipMemcpyHostToDevice);
	cudaCheckError("cudaMemcopy to device IN tensor");

	// IN DATA

	float *d_in_data;
	long in_data_size = sizeof(*d_in_data) * in.getSize().x * in.getSize().y
			* in.getSize().z;

	//printf("sizeof(in)= %lu , in_data_size = %lu \n", sizeof(in), in_data_size);

	hipMalloc((void **) &d_in_data, in_data_size);
	cudaCheckError("hipMalloc IN tensor data");

	hipMemcpy(d_in_data, in.data, in_data_size, hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy to device IN tensor data");

	hipMemcpy(&(d_in->data), &d_in_data, sizeof(d_in->data),
			hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy Binding pointers of IN tensor data");

	// Copy weights

	h_weights = &weights;
	long weights_mem_size = sizeof(weights);

	//printf("sizeof(weights) == weights_mem_size => %lu == %lu \n",sizeof(weights), weights_mem_size);

	if (h_weights == NULL) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	hipMalloc((void **) &d_weights, weights_mem_size);
	cudaCheckError("hipMalloc Weights tensor");

	hipMemcpy(d_weights, h_weights, weights_mem_size, hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy to device Weights tensor");

	// Weights DATA

	float *d_weights_data;
	long weights_data_size = sizeof(*d_weights_data) * weights.getSize().x
			* weights.getSize().y * weights.getSize().z;

	hipMalloc((void **) &d_weights_data, weights_data_size);
	cudaCheckError("hipMalloc Weights tensor data");

	hipMemcpy(d_weights_data, weights.data, weights_data_size,
			hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy to device Weights tensor data");

	hipMemcpy(&(d_weights->data), &d_weights_data, sizeof(d_weights->data),
			hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy Binding pointers of Weights tensor data");

	// Reserve input memory space

	h_input = &input[0];
	long input_mem_size = sizeof(input[0]) * input.size();

	//printf("input_mem_size : %lu \n", input_mem_size);

	hipMalloc((void **) &d_input, input_mem_size);
	cudaCheckError("hipMalloc Input array");

	hipMemcpy(d_input, h_input, input_mem_size, hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy to device Input array");

	// Reserve memory space for OUT

	h_out = &out;
	int out_mem_size = sizeof(out);

	hipMalloc((void **) &d_out, out_mem_size);
	cudaCheckError("hipMalloc Out tensor");

	hipMemcpy(d_out, h_out, out_mem_size, hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy to device Out tensor");

	// Out DATA

	float *d_out_data;
	long out_data_size = sizeof(*d_out_data) * h_out->getSize().x
			* h_out->getSize().y * h_out->getSize().z;

	//printf("out_data_size : %lu \n", out_data_size);

	hipMalloc((void **) &d_out_data, out_data_size);
	cudaCheckError("hipMalloc Out tensor data");

	hipMemcpy(d_out_data, h_out->data, out_data_size, hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy to device Out tensor data");

	hipMemcpy(&(d_out->data), &d_out_data, sizeof(d_out->data),
			hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy Binding pointers of Out tensor data");

	//printf("h_out[0,0,0]: %f, \n", h_out->get(0, 0, 0));
	// TODO
	//printf("Tensor in: \n");
	//print_tensor(*h_in);

	// TODO remove
	//printf("input size: %lu input 0: %f \n", input.size(), input[0]);

	//printf("h_input 0: %f \n", *h_input);

	//printf("out: %i, %i, %i \n", out.getSize().x, out.getSize().y,out.getSize().z);

	//printf("h_out: %i, %i, %i \n", h_out->getSize().x, h_out->getSize().y,h_out->getSize().z);

	// Lanzar KERNEL

	Logger::debug("CUDA kernel launch with %d blocks of %d threads. Total: %i\n",
			blocksPerGrid, threadsPerBlock, totalThreads);

	activate_cuda<<<blocksPerGrid, threadsPerBlock>>>(d_in, d_weights, d_input,
			d_out);

	hipDeviceSynchronize();

	cudaCheckError("Launch kernel");

	// get input array

	hipMemcpy(h_input, d_input, input_mem_size, hipMemcpyDeviceToHost);
	cudaCheckError("hipMemcpy to host Input array");

	hipFree(d_input);
	cudaCheckError("hipFree Input array");

	// Get Out DATA

	hipMemcpy(h_out->data, d_out_data, out_data_size, hipMemcpyDeviceToHost);
	cudaCheckError("hipMemcpy to host Out tensor data");

	hipFree(d_out);
	cudaCheckError("hipFree Out tensor");

	// Free host memory

	hipDeviceReset();

	cudaCheckError("hipDeviceReset");
	// TODO remove
	//printf("cuda out: %i, %i, %i \n", h_out->getSize().x, h_out->getSize().y,h_out->getSize().z);
	//printf("cuda out[0,0,0]: %f, \n", h_out->get(0, 0, 0));
	//print_tensor(*h_out);
}

