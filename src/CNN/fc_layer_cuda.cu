#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <vector>
#include <algorithm>   // std::min

#include <math.h>
#include <float.h>
#include <string.h>

#include "gradient_t.h"
#include "layer_t.h"
#include "optimization_method.h"

#include "fc_layer_cuda.h"

#include "CUDA/utils.cuh"

void cudaActivate(tensor_t<float> in, tensor_t<float> weights,
		std::vector<float> &input, tensor_t<float> &out);

__host__ __device__ float activator_function(float x) {
	//return tanhf( x );
	float sig = 1.0f / (1.0f + exp(-x));
	return sig;
}

__host__ void fc_layer_cuda_t::activate(tensor_t<float>& in) {
	this->in = in;
	//activate();

	cudaActivate(this->in, this->weights, this->input, this->out);

	// TODO
	exit(EXIT_SUCCESS);

}

/**
 * CUDA Kernel Device code
 *
 */
/*****************************************************************************/

__device__ float& get(tensor_t<float> *t, int _x, int _y, int _z) {
	assert(_x >= 0 && _y >= 0 && _z >= 0);
	assert(_x < t->size.x && _y < t->size.y && _z < t->size.z);

	return t->data[_z * (t->size.x * t->size.y) + _y * (t->size.x) + _x];
}

__device__ void set(tensor_t<float> *t, int _x, int _y, int _z, float value) {
	assert(_x >= 0 && _y >= 0 && _z >= 0);
	assert(_x < t->size.x && _y < t->size.y && _z < t->size.z);

	t->data[_z * (t->size.x * t->size.y) + _y * (t->size.x) + _x] = value;
}

__global__ void activate_cuda(tensor_t<float> *d_in, tensor_t<float> *d_weights,
		float *d_input, tensor_t<float> *d_out) {

	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	int i = ((int) index / (d_in->size.y * d_in->size.z)) % d_in->size.x;
	int j = ((int) index / d_in->size.z) % d_in->size.y;
	int k = index % d_in->size.z;

	printf("index: %i  (x, y, z)=(%i, %i, %i)  (i, j, k)=(%i, %i, %i) \n",
	 index, d_in->size.x, d_in->size.y, d_in->size.z, i, j, k);

	for (int n = 0; n < d_out->size.x; n++) {
		float inputv = 0;

		// map
		int m = k * (d_in->size.x * d_in->size.y) + j * (d_in->size.x) + i;

		inputv += get(d_in, i, j, k) * get(d_weights, m, n, 0);

		//printf("inputv: %f \n", inputv);

		*(d_input + n) = inputv;

		set(d_out, n, 0, 0, activator_function(inputv));
	}

}

/******************************************************************************
 * Host main routine
 */
void cudaActivate(tensor_t<float> in, tensor_t<float> weights,
		std::vector<float> &input, tensor_t<float> &out) {

	int blocksPerGrid, threadsPerBlock;
	int totalThreads;
	tensor_t<float> *h_in, *d_in;
	tensor_t<float> *h_weights, *d_weights;
	float *h_input, *d_input;
	tensor_t<float> *h_out, *d_out;

	// Get device info
	int dev = 0;
	hipSetDevice(dev);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);

	int requiredThreads = in.size.x * in.size.y * in.size.z;

	blocksPerGrid = std::min(deviceProp.multiProcessorCount, 2);

	threadsPerBlock = std::min(deviceProp.maxThreadsPerBlock,
			requiredThreads / blocksPerGrid);

	totalThreads = blocksPerGrid * threadsPerBlock;

	h_in = &in;
	int in_mem_size = sizeof(in);

	if (h_in == NULL) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	hipMalloc((void **) &d_in, in_mem_size);
	cudaCheckError("hipMalloc IN tensor");

	hipMemcpy(d_in, h_in, in_mem_size, hipMemcpyHostToDevice);
	cudaCheckError("cudaMemcopy to device IN tensor");

	// IN DATA

	float *d_in_data;
	long in_data_size = sizeof(*d_in_data) * in.getSize().x * in.getSize().y
			* in.getSize().z;

	//printf("sizeof(in)= %lu , in_data_size = %lu \n", sizeof(in), in_data_size);

	hipMalloc((void **) &d_in_data, in_data_size);
	cudaCheckError("hipMalloc IN tensor data");

	hipMemcpy(d_in_data, in.data, in_data_size, hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy to device IN tensor data");

	hipMemcpy(&(d_in->data), &d_in_data, sizeof(d_in->data),
			hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy Binding pointers of IN tensor data");

	// Copy weights

	h_weights = &weights;
	long weights_mem_size = sizeof(weights);

	//printf("sizeof(weights) == weights_mem_size => %lu == %lu \n",sizeof(weights), weights_mem_size);

	if (h_weights == NULL) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	hipMalloc((void **) &d_weights, weights_mem_size);
	cudaCheckError("hipMalloc Weights tensor");

	hipMemcpy(d_weights, h_weights, weights_mem_size, hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy to device Weights tensor");

	// Weights DATA

	float *d_weights_data;
	long weights_data_size = sizeof(*d_weights_data) * weights.getSize().x
			* weights.getSize().y * weights.getSize().z;

	hipMalloc((void **) &d_weights_data, weights_data_size);
	cudaCheckError("hipMalloc Weights tensor data");

	hipMemcpy(d_weights_data, weights.data, weights_data_size,
			hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy to device Weights tensor data");

	hipMemcpy(&(d_weights->data), &d_weights_data, sizeof(d_weights->data),
			hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy Binding pointers of Weights tensor data");

	// Reserve input memory space

	h_input = &input[0];
	long input_mem_size = sizeof(input[0]) * input.size();

	//printf("input_mem_size : %lu \n", input_mem_size);

	hipMalloc((void **) &d_input, input_mem_size);
	cudaCheckError("hipMalloc Input array");

	hipMemcpy(d_input, h_input, input_mem_size, hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy to device Input array");

	// Reserve memory space for OUT

	h_out = &out;
	int out_mem_size = sizeof(out);

	hipMalloc((void **) &d_out, out_mem_size);
	cudaCheckError("hipMalloc Out tensor");

	hipMemcpy(d_out, h_out, out_mem_size, hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy to device Out tensor");

	// Out DATA

	float *d_out_data;
	long out_data_size = sizeof(*d_out_data) * h_out->getSize().x
			* h_out->getSize().y * h_out->getSize().z;

	//printf("out_data_size : %lu \n", out_data_size);

	hipMalloc((void **) &d_out_data, out_data_size);
	cudaCheckError("hipMalloc Out tensor data");

	hipMemcpy(d_out_data, h_out->data, out_data_size, hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy to device Out tensor data");

	hipMemcpy(&(d_out->data), &d_out_data, sizeof(d_out->data),
			hipMemcpyHostToDevice);
	cudaCheckError("hipMemcpy Binding pointers of Out tensor data");

	// Lanzar KERNEL

	Logger::debug("CUDA kernel launch with %d blocks of %d threads. Total: %i\n",
			blocksPerGrid, threadsPerBlock, totalThreads);

	activate_cuda<<<blocksPerGrid, threadsPerBlock>>>(d_in, d_weights, d_input,
			d_out);

	hipDeviceSynchronize();

	cudaCheckError("Launch kernel");

	// get input array

	hipMemcpy(h_input, d_input, input_mem_size, hipMemcpyDeviceToHost);
	cudaCheckError("hipMemcpy to host Input array");

	hipFree(d_input);
	cudaCheckError("hipFree Input array");

	// Get Out DATA

	hipMemcpy(h_out->data, d_out_data, out_data_size, hipMemcpyDeviceToHost);
	cudaCheckError("hipMemcpy to host Out tensor data");

	hipFree(d_out);
	cudaCheckError("hipFree Out tensor");

	// Free host memory

	hipDeviceReset();

	cudaCheckError("hipDeviceReset");
	// TODO remove
	//printf("cuda out: %i, %i, %i \n", h_out->getSize().x, h_out->getSize().y,h_out->getSize().z);
	//printf("cuda out[0,0,0]: %f, \n", h_out->get(0, 0, 0));
	//print_tensor(*h_out);
}

