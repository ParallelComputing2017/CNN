
#include <string>

#include "utils.cuh"


void cudaCheckError(std::string msg) {
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to: %s (error code %s)!\n", msg.c_str(),
				hipGetErrorString(err));
		exit (EXIT_FAILURE);
	}
}

