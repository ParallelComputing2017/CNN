#include "hip/hip_runtime.h"
/*
 * cudaConvLayer.cu
 *
 *  Created on: 12/11/2017
 *      Author: sebas
 */

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include "cudaConvLayer.h"
#include "cudaTensor.cuh"

__device__ point_t map_to_input(int stride, point_t out, int z) {
	out.x *= stride;
	out.y *= stride;
	out.z = z;
	return out;
}

__global__ void convolutionKernel(tensor_t<float> *in, tensor_t<float> *kernel,
		int *filterIdx, int *stride, tensor_t<float> *out) {

	for (int x = 0; x < out->size.x; x++) {
		for (int y = 0; y < out->size.y; y++) {
			point_t p_out = (point_t ) { x, y, 0 };
			point_t mapped = map_to_input((*stride), p_out, 0);
			float sum = 0;
			for (int i = 0; i < kernel->size.x; i++) {
				for (int j = 0; j < kernel->size.y; j++) {
					for (int z = 0; z < in->size.z; z++) {
						float f = cudaTensor::get(kernel, i, j, z);
						float v = cudaTensor::get(in,mapped.x + i, mapped.y + j, z);
						sum += f * v;
					}
				}
			}
			cudaTensor::set(out, x, y, *filterIdx, sum);
		}
	}
}

void cudaConvolution(tensor_t<float> *in, tensor_t<float> *kernel,
		int *filterIdx, int *stride, tensor_t<float> *out) {

	int blocksPerGrid, threadsPerBlock;
	int totalThreads;

	// Get device info
	int dev = 0;
	hipSetDevice(dev);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);

	int requiredThreads = 1;

	blocksPerGrid = std::min(deviceProp.multiProcessorCount, 1);

	threadsPerBlock = std::min(deviceProp.maxThreadsPerBlock,
			requiredThreads / blocksPerGrid);

	totalThreads = blocksPerGrid * threadsPerBlock;

	// IN

	cudaTensor inTensor(in);
	inTensor.hostToDevice();

	// Kernel
	cudaTensor kernelTensor(kernel);
	kernelTensor.hostToDevice();

	// Out

	cudaTensor outTensor(out);
	outTensor.hostToDevice();

	// Filter

	int* d_filter;
	long filter_mem_size = sizeof(int);

	hipMalloc((void **) &d_filter, filter_mem_size);
	cudaCheckError();

	hipMemcpy(d_filter, filterIdx, filter_mem_size, hipMemcpyHostToDevice);
	cudaCheckError();

	// Stride

	int* d_stride;
	long stride_mem_size = sizeof(int);

	hipMalloc((void **) &d_stride, stride_mem_size);
	cudaCheckError();

	hipMemcpy(d_stride, stride, stride_mem_size, hipMemcpyHostToDevice);
	cudaCheckError();

	// Launch KERNEL

	Logger::debug(
			"CUDA kernel launch with %d blocks of %d threads. Total: %i\n",
			blocksPerGrid, threadsPerBlock, totalThreads);

	convolutionKernel<<<blocksPerGrid, threadsPerBlock>>>(
			inTensor.devicePointer(), kernelTensor.devicePointer(), d_filter,
			d_stride, outTensor.devicePointer());

	hipDeviceSynchronize();
	cudaCheckError();

	// get out

	outTensor.deviceToHost();

	// Free device memory

	inTensor.deviceFree();
	kernelTensor.deviceFree();
	outTensor.deviceFree();

	hipFree(d_filter);
	cudaCheckError();

	hipFree(d_stride);
	cudaCheckError();

	hipDeviceReset();
	cudaCheckError();

	// Free host memory
}

void CudaConvLayer::activate(tensor_t<float>& in) {

	this->in = in;

	for (int filter = 0; filter < filters.size(); filter++) {
		tensor_t<float> *kernel = &filters[filter];

		tensor_t<float> *pIn = &in;
		int *pFilter = &filter;
		int iStride = (int) stride;
		int *pStride = &iStride;
		tensor_t<float> *pOut = &out;

		cudaConvolution(pIn, kernel, pFilter, pStride, pOut);

		// TODO
		//exit(EXIT_SUCCESS);
	}
}

